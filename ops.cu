#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>

at::Tensor op_cuda(at::Tensor Arow, at::Tensor Acol, at::Tensor Aval, at::Tensor b, at::Tensor out){
    size_t size_qr = 0;
    size_t size_internal = 0;
    void *buffer_qr = NULL;
    hipsolverSpHandle_t cusolverH = NULL;
    csrqrInfo_t info = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
        
    cusolver_status = hipsolverSpCreate(&cusolverH);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cusparse_status = hipsparseCreateMatDescr(&descrA); 
    assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE); // base-1

    cusolver_status = cusolverSpCreateCsrqrInfo(&info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    
    const int bs = b.size(0);
    const int m = b.size(1);
    const int nA = Aval.size(1);
    auto aval_pointer = Aval.data_ptr<double>();
    auto arow_pointer = Arow.data_ptr<int>();
    auto acol_pointer = Acol.data_ptr<int>();    
    auto b_pointer = b.data_ptr<double>();
    auto out_pointer = out.data_ptr<double>();

    cusolver_status = cusolverSpXcsrqrAnalysisBatched(
        cusolverH, m, m, nA,
        descrA, arow_pointer, acol_pointer, info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cusolver_status = cusolverSpDcsrqrBufferInfoBatched(
        cusolverH, m, m, nA,
        descrA, aval_pointer, arow_pointer, acol_pointer,
        bs, info, &size_internal, &size_qr);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    
    hipMalloc((void**)&buffer_qr, size_qr);

    cusolver_status = cusolverSpDcsrqrsvBatched(
        cusolverH, m, m, nA,
        descrA, aval_pointer, arow_pointer, acol_pointer,
        b_pointer, out_pointer, bs, info, buffer_qr);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    return out;
}
